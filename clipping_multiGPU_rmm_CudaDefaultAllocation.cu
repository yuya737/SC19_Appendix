
#include <algorithm>
#include <iostream>
#include <numeric>
#include <array>
#include <vector>
#include <stdlib.h>
#include <random>
#include <thread>

#include <thrust/reduce.h>
#include <thrust/count.h>
#include <thrust/remove.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/system/cuda/execution_policy.h>
#include "hip/hip_runtime.h"
#include ""

#include <thread>

#include "cClipping.h"
#include "cTimer.h"
#include "thrust_rmm_allocator.h"

cTimer timer;

typedef rmm::device_vector<float>::iterator IterFloat;
typedef rmm::device_vector<int>::iterator IterInt;

#define MB (1024*1024)
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/**
    Clips the input vector given the input-defined plane 

	@param 
		posIn: input vector to be clipped
		normal: normal of the plane with which posIn will be clipped
		d: the point along with the nomral with which posIn will be clipped
*/

void clip (rmm::device_vector<float> *posIn, float *normal, float d)
{
	plane_clippingPDBver2 clip	(normal, d);

	strided_range<IterFloat> X		( posIn->begin()  , posIn->end(), 4);
	strided_range<IterFloat> Y		( posIn->begin()+1, posIn->end(), 4);
	strided_range<IterFloat> Z		( posIn->begin()+2, posIn->end(), 4);
	strided_range<IterFloat> W		( posIn->begin()+3, posIn->end(), 4);

	size_t new_size = thrust::remove_if( thrust::make_zip_iterator ( thrust::make_tuple( X.begin(), Y.begin(), Z.begin(), W.begin() )),
			   	   	   	   	   	   	   	 thrust::make_zip_iterator ( thrust::make_tuple( X.end(),Y.end(), Z.end(), W.end() )),
			   	   	   	   	   	   	   	 clip )
	                      	  	  	  	- thrust::make_zip_iterator(thrust::make_tuple(X.begin(), Y.begin(), Z.begin(), W.begin() ));

	// Resize the input vector to appropriate size
	posIn->resize(new_size*4);
}

/**
    Copy vec to d_vec on the given thread

	@param 
		devId: stream ID used to copy the vectors
		vec: vector to be copied from
		d_vec: vector to be copied to
*/

void memCopyHtoD (int devId, std::vector<float> *vec, rmm::device_vector<float> *d_vec )
{
	gpuErrchk(hipSetDevice(devId));
	*d_vec = *vec;
}

/**
    Copy vec to d_vec on the given thread

	@param 
		devId: stream ID used to copy the vectors
		vec: vector to be copied from
		d_vec: vector to be copied to
*/

void memCopyDtoH (int devId, std::vector<float> *vec, rmm::device_vector<float> *d_vec )
{
	gpuErrchk(hipSetDevice(devId));
	vec->resize(d_vec->size());
	thrust::copy(d_vec->begin(), d_vec->end(), vec->begin());
}

/**
    Run clipping on given thread

	@param 
		devId: stream ID used to run the clipping
		d_pos: vector to be clipped
		normal: normal of the plane with which d_pos will be clipped
		d: the point along with the nomral with which d_pos will be clipped
*/
void launch (int devId, rmm::device_vector<float> *d_pos, float *normal, float d)
{
	gpuErrchk(hipSetDevice(devId));
	clip (d_pos, normal, d);

}

int main(int argc, char *argv[])
{
    size_t sx, sy, sz;
    int numStreams, i, j;

    int deviceCount = 0;
    unsigned int iter = 0;

	double elapsed = 0.0;
	double totalElapsed = 0.0;
	double htodElapsed = 0.0;
	double dtohElapsed = 0.0;
	double transferElapsed = 0.0;
	double computeElapsed = 0.0;

    // This willl be used to generate plane's normals randomly
    // between -1 to 1
    std::mt19937 rng(time(NULL));
	std::uniform_real_distribution<float> gen(-1.0, 1.0);
	
    // plane defined by normal and D
    float normal[3], d = 0.5f;

    if (argc < 5){
        std::cout << "Usage: clipping x_size y_size z_size iterations" << std::endl;
        return 1;
    }
    sx = std::stoll (std::string(argv[1]));
    sy = std::stoll (std::string(argv[2]));
    sz = std::stoll (std::string(argv[3]));
    iter = std::stoi (std::string(argv[4]));

	size_t numParticles = sx*sy*sz;
	
	// Retrieve the number of streams/devices we can use
    gpuErrchk(hipGetDeviceCount(&deviceCount));
    numStreams = deviceCount;
    size_t numParticlesPerThread = sx*sy*sz/numStreams;

    std::cout << "========\n";
	std::cout << "Domain size is " << sx << " x " << sy << " x " << sz << " = " << numParticles << " particles" << std::endl;
	std::cout << "Size MB " << (sizeof(float) * numParticles * 4.0) / MB <<std::endl;
	std::cout << "Num. Devices " << deviceCount << std::endl;
	std::cout << "Particles per device " << numParticlesPerThread << std::endl;
	std::cout << "Size MB per device " << (sizeof(float) * numParticlesPerThread * 4.0) / MB <<std::endl;

	// Define array of vectors and threads to be used 
    std::thread thread[numStreams];
	std::vector <float> pos[numStreams];
	std::vector <float> posOut[numStreams];
	rmm::device_vector<float> d_pos[numStreams];

	std::cout << "Generated particles...\n";
	
	// Types of allocations:
	// CudaDefaultAllocation
	// PoolAllocation
	// CudaManagedMemory

	rmmOptions_t options{rmmAllocationMode_t::CudaDefaultAllocation, 0, true};
	rmmInitialize(&options);
	
	// Timer to record time taken to initialize dataset
	timer.reset();

	// Launch threads and join
	for (i=0;i<numStreams;i++)
	{
		size_t szMin =  i*(sz/numStreams);
		size_t szMax = (i+1)*(sz/numStreams);
		thread[i] =std::thread (initDatasetChunk, &pos[i],  sx, sy, szMin, szMax);
	}
	for(i = 0; i < numStreams; i++)
	{
		thread[i].join ();
	}
	std::cout << "in " << timer.getElapsedMilliseconds() << " ms\n";


	for(j=0;j<iter;j++)
	{
		// Generating plane's normals randomly
		// between -1 to 1
		normal[0] = gen(rng);
		normal[1] = gen(rng);
		normal[2] = gen(rng);

		// Copy H to D
		timer.reset ();
		for (i=0;i<numStreams;i++)
		{
			thread[i] =std::thread (memCopyHtoD, i, &pos[i], &d_pos[i]);
		}
		for(i = 0; i < numStreams; i++)
		{
			thread[i].join ();
		}
		elapsed = timer.getElapsedMilliseconds();
		std::cout << "H to D: " << elapsed << " ms\n";
		htodElapsed+=elapsed;
		transferElapsed+=elapsed;

		// launch the clip kernel
		timer.reset ();
		for(i = 0; i < numStreams; i++)
		{
			thread[i] =std::thread (launch, i, &d_pos[i], normal, d );
		}
		for(i = 0; i < numStreams; i++)
		{
			thread[i].join ();
		}
		elapsed = timer.getElapsedMilliseconds();
		//std::cout << "Clipping: " << elapsed << " ms\n";
		computeElapsed += elapsed;

		// Copy D to H
		timer.reset ();
		for (i=0;i<numStreams;i++)
		{
			thread[i] =std::thread (memCopyDtoH, i, &posOut[i], &d_pos[i]);
		}
		for(i = 0; i < numStreams; i++)
		{
			thread[i].join ();
		}
		elapsed = timer.getElapsedMilliseconds();
		//std::cout << "D to H: " << elapsed << " ms\n";
		dtohElapsed+=elapsed;
		transferElapsed+=elapsed;
	}
	std::cout << "--------\n";
	totalElapsed = computeElapsed + transferElapsed;
	std::cout << "H to D Avg time (ms) after " << iter << " iterations " << htodElapsed / iter << std::endl;
	std::cout << "D to H Avg time (ms) after " << iter << " iterations " << dtohElapsed / iter << std::endl;
	std::cout << "Transfers Avg time (ms) after " << iter << " iterations " << transferElapsed / iter << std::endl;
	std::cout << "Compute Avg time (ms) after " << iter << " iterations " << computeElapsed / iter << std::endl;
	std::cout << "Total Avg time (ms) after " << iter << " iterations " << totalElapsed / iter << std::endl;

    return 0;

}



